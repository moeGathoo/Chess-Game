#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <chrono>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""

// Kernel function to add the elements of two arrays
__global__ void add(int n, float* x, float* y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n)
        y[index] = x[index] * 10;
}

int main(void)
{
    int N = 20;
    float* x, * y;
    float* host = new float[N * sizeof(float)];
    float* out = new float[N * sizeof(float)];

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMalloc(& x, N * sizeof(float));
    hipMalloc(& y, N * sizeof(float));
    std::cout << N << std::endl;

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        host[i] = i;
    }

    hipMemcpy(&x, &host, N * sizeof(float), hipMemcpyHostToDevice);

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    hipEventRecord(begin, 0);
    add << <numBlocks, blockSize>> > (N, x, y);
    hipEventRecord(end, 0);
    hipEventSynchronize(end);
    hipMemcpy(&out, &y, N * sizeof(float), hipMemcpyDeviceToHost);
    float time = 0;
    hipEventElapsedTime(&time, begin, end);
    for (int i = 0; i < N; i++)
        std::cout << out[i] << std::endl;
    std::cout << "Kernel run time: " << time << std::endl;

    auto start = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < N; i++) {
        out[i] = host[i] * 10;
    }
    auto stop = std::chrono::high_resolution_clock::now();
    long long timeTaken = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start).count();
    std::cout << "CPU Time: " << timeTaken << std::endl;

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    std::cout << "added" << std::endl;
    //std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
    free(host);

    return 0;
}